#include "hip/hip_runtime.h"
#include "fluidsCuda.h"
#include "fluidsCuda.cu"
using namespace std;

int main(int argc, char *argv[])
{
	unsigned int simWidth = 128;	// x divisions
	unsigned int simHeight = 128;// y divisions
									// note: 6mx6m with 256x256 yields ~square inch resolution
	float xRange = 6.0; 			// meters
	float yRange = 6.0;				// meters
	float frameVel = 7.0;			// speed of object, right to left

	float dt = 0.001;

	int colorIndexCount = simWidth * simHeight * 4;
	int realIndexCount = simWidth * simHeight;
	cout <<"	frameVel will cover "<<frameVel*dt/(xRange/simWidth)<<" quads" <<endl;

			/////////*** Quad Vertices and Color data, 4x as large as real data ***/////////

	float3 *quadPoints = (float3*)malloc(colorIndexCount*sizeof(float3));
	float3 *colors = (float3*)malloc(colorIndexCount*sizeof(float3));

	int idxRow = 0;
	int idxCol = 0;
	float halfHeight = 0.0 ; // (float)0.5/simHeight;
	float halfWidth = 0.0 ; //(float)0.5/simWidth;
	for (int i = 0; i < colorIndexCount; i++){
		if(i%4==0){ //top left of a quad
			quadPoints[i].x = 2.0*(float)idxCol/simWidth - 1.0f + halfWidth;
			quadPoints[i].y = 1.0f - 2.0*(float)idxRow/simHeight - halfHeight;
			quadPoints[i].z = 0.0f;
		}
		if(i%4==1){ //bottom left of a quad
			quadPoints[i].x = 2.0*(float)idxCol/simWidth - 1.0f + halfWidth;
			quadPoints[i].y = 1.0f - 2.0*(float)(idxRow+1)/simHeight - halfWidth;
			quadPoints[i].z = 0.0f;
		}
		if(i%4==2){ //bottom right of a quad
			quadPoints[i].x = 2.0*(float)(idxCol+1)/simWidth - 1.0f + halfWidth;
			quadPoints[i].y = 1.0f - 2.0*(float)(idxRow+1)/simHeight - halfHeight;
			quadPoints[i].z = 0.0f;
		}
		if(i%4==3){ //top right of a quad
			quadPoints[i].x = 2.0*(float)(idxCol+1)/simWidth - 1.0f + halfWidth;
			quadPoints[i].y = 1.0f - 2.0*(float)idxRow/simHeight - halfHeight;
			quadPoints[i].z = 0.0f;
			idxCol++;
		}
		if(idxCol == simWidth){ // row of quads done
			idxCol = 0;
			idxRow++;
		}
	}

	for (int i = 0; i < colorIndexCount; i++){
		colors[i].x = 0.0f;
		colors[i].y = 0.0f; 
		colors[i].z = 0.0f;
	}
			////////*** "Real" Vertices and Velocity data, the size of simDimensions ***/////////

			// I don't bother w/ cuda malloc on host because I don't plan on data transfers

	size_t latticeSize = realIndexCount*sizeof(float2);
	float2 *devPositions; float2 *devVelocities;
	float2 *devVelocities2; 
	float2 *positions = (float2*)malloc(latticeSize);
	float2 *velocities = (float2*)malloc(latticeSize);
	checkCuda(hipMalloc((void**)&devPositions, latticeSize));
	checkCuda(hipMalloc((void**)&devVelocities, latticeSize));
	checkCuda(hipMalloc((void**)&devVelocities2, latticeSize));

	for (int row = 0; row < simHeight; row++){
		for (int col = 0; col < simWidth; col++){
			positions[row*simWidth+col].x = xRange*(float)col/simWidth - xRange/2.0;
			positions[row*simWidth+col].y = yRange/2.0 - yRange*(float)row/simHeight;
			velocities[row*simWidth+col].x = frameVel;
			/*
			if(row*simWidth+col > simWidth*(simHeight/2) && row*simWidth+col < simWidth*(simHeight/2)+20)
				velocities[row*simWidth+col].x = frameVel + 10;
			*/

			velocities[row*simWidth+col].y = 0.0;
		}
	}
	for(int i = 0; i < 2; i++){
		velocities[simWidth * simHeight/2 + i*simWidth + simWidth/2].x = 15.0;
	}
	
	float4 boundaries = make_float4(positions[0].x, positions[simWidth*simHeight-1].y,
									positions[simWidth*simHeight-1].x, positions[0].y);
	float dr = xRange/simWidth;
	cout<<"	dR resolution: "<<dr<<" Meters"<<endl;

	checkCuda(hipMemcpy(devPositions, positions, latticeSize, hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(devVelocities, velocities, latticeSize, hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(devVelocities2, velocities, latticeSize, hipMemcpyHostToDevice));

			/////////*** GLEW Initialization, quarter window ***/////////

	if (!glfwInit()) {
		fprintf(stderr, "ERROR: could not start GLFW3\n");
		return 1;
	} 
	GLFWwindow* window = glfwCreateWindow(1920/(1.5), 1080/(1.5), "Cuda Fluid Dynamics", NULL, NULL);
	glfwSetWindowPos(window, 1920/2, 0);
	if (!window) {
		fprintf(stderr, "ERROR: could not open window with GLFW3\n");
		glfwTerminate();
		return 1;
	}
	glfwMakeContextCurrent(window);
	glewExperimental = GL_TRUE;
	glewInit();

			/////////*** Create position and color vertex buffer objects ***/////////

		// Quad Position Buffer Array

	GLuint pointsVBO = 0;
	glGenBuffers(1, &pointsVBO);
	glBindBuffer(GL_ARRAY_BUFFER, pointsVBO);
	glBufferData(GL_ARRAY_BUFFER, colorIndexCount*sizeof(float3), quadPoints, GL_STATIC_DRAW);

		// Color Buffer Array

	GLuint colorsVBO = 0;
	glGenBuffers(1, &colorsVBO);
	glBindBuffer(GL_ARRAY_BUFFER, colorsVBO);
	glBufferData(GL_ARRAY_BUFFER, colorIndexCount*sizeof(float3), colors, GL_DYNAMIC_DRAW);
	struct hipGraphicsResource *cudaColorResource;
	checkCuda(hipGraphicsGLRegisterBuffer(&cudaColorResource, colorsVBO, cudaGraphicsMapFlagsNone));

		// vao binding

	GLuint vertexArray = 0;
	glGenVertexArrays(1, &vertexArray);
	glBindVertexArray(vertexArray);

	glBindBuffer(GL_ARRAY_BUFFER, pointsVBO);
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 0, NULL);

	glBindBuffer(GL_ARRAY_BUFFER, colorsVBO);
	glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 0, NULL);

	glEnableVertexAttribArray(0);
	glEnableVertexAttribArray(1);

			/////////*** Create shader program ***/////////

	GLuint shaderProgram = glCreateProgram();
	glInitShaders(vertexShaderText, fragmentShaderText, shaderProgram);	
	
			/////////*** Draw Loop ***/////////
	
	hipEvent_t start, stop; 
	float fpsTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	dim3 tpbColor(0, 0);
	dim3 tpbLattice(0, 0);
	dim3 blocks(0, 0);
	initThreadDimensions(simWidth, simHeight, tpbColor, tpbLattice, blocks);
	cout<<"	Calling with Boundaries: "<<boundaries.x<<" "<<boundaries.y<<" "<<boundaries.z<<" "<<boundaries.w<<endl;

	bool test = false;
	//int j = 0;
	while(!glfwWindowShouldClose(window) && !test) {
		hipEventRecord(start, 0);
		//sleep(1);

			// Run all CUDA kernels including colorization of the linked resource
		/*
		for(int i = simWidth*simHeight/2; i < simWidth*simHeight/2 + 30; i++)
			cout<<"vel: "<< velocities[i].x <<endl;
			if(j==3)
				test = true;
		cout<<endl;
		j++;*/

		runCuda(&cudaColorResource, 
				devPositions, devVelocities, devVelocities2,
				boundaries, dt, dr, 
				tpbColor, tpbLattice, blocks, 
				simWidth, simHeight);
		// TESTING 
		checkCuda(hipMemcpy(velocities, devVelocities2, latticeSize, hipMemcpyDeviceToHost));
				glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
		glUseProgram(shaderProgram);
		glBindVertexArray(vertexArray);
		glDrawArrays(GL_QUADS, 0, colorIndexCount);

		glfwSwapBuffers(window);
  		glfwPollEvents();

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&fpsTime, start, stop);
		char title[256];
		sprintf(title, "CudaFluidDynamics: %12.2f fps", 1.0f/(fpsTime/1000.0f));
		glfwSetWindowTitle(window, title);

		

		if(glfwGetKey(window, GLFW_KEY_ESCAPE)) {
			glfwSetWindowShouldClose(window, 1);
		}
	}

	glDeleteBuffers(1, &pointsVBO);
	glDeleteBuffers(1, &colorsVBO);
	glDeleteVertexArrays(1, &vertexArray);
	hipFree(devVelocities);
	hipFree(devPositions);
	hipFree(devVelocities2);
	free(quadPoints);
	free(velocities);
	free(positions);
	free(colors);
	glfwTerminate();
	return 0;
}
