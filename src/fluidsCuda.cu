#include "hip/hip_runtime.h"
#include "fluidsCuda.h"

			/* Kernel to operate on color data by quadIdx, 
			 * might become null as color will be represented
			 * by velocity data directly in the future */

__global__ void
velToColor(float3 *colors,
		   float3 *colorMap,
		   float2 *__restrict__ newVel,
		   dim3 blocks,
		   unsigned int simWidth,
		   unsigned int simHeight)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	int quadIdx = x + simWidth*y;
	float magVel = sqrt(newVel[quadIdx].x * newVel[quadIdx].x + newVel[quadIdx].y * newVel[quadIdx].y);
	int map10_256 = (int)(magVel/7.0 * 256);
	if(map10_256 > 256) { map10_256 = 256; }


	for(int i = 0; i < 4; i++){
		colors[4*quadIdx+i] = colorMap[map10_256];
	}

	/*
	for(int i = 0; i < 4; i++){
		if (newVel[quadIdx].x < 0){
			colors[4*quadIdx+i].x = 1.0;
		}
	}*/
}

			/* Obstruct method, simply zero's velocities 
			 * at chosen object locations. This is probably 
			 * not what I want in the end as their should be
			 * some pressure conditions at the edge or something TODO */

__global__ void
Obstruct(int *__restrict__ obstructed,
		 float2 *__restrict__ oldVel)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	int obstruct = obstructed[x];
	oldVel[obstruct] = make_float2(0.0, 0.0);
}

			/* Diffusion method, uses iterative jacobi method 
			 * to approximate solutions to poisson's eqn in 
			 * diffusion. Should be called for a number of iterations */

__global__ void
Diffuse(float2 *__restrict__ positions,
		float2 *__restrict__ oldVel,
		float2 *__restrict__ newVel,
		float dt,
		float dr,
		float viscosity,
		unsigned int simWidth,
		unsigned int simHeight)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	float2 Vel = oldVel[y*simWidth + x];
	float2 TVel;
	float2 LVel;
	float2 BVel;
	float2 RVel;
	float alpha = dr * dr / (viscosity * dt);

	if (x!=0 && y!=0 && x!=simWidth-1 && y!=simHeight-1)
	{
		TVel = oldVel[(y-1)*simWidth + x];
		LVel = oldVel[(y*simWidth) + x - 1];
		BVel = oldVel[(y+1)*simWidth + x];
		RVel = oldVel[(y*simWidth) + x + 1];

		newVel[y*simWidth + x] = JacobiInstance(TVel, LVel, 
												BVel, RVel,
												alpha, Vel);
	}
}

			/* Advection method, utilizes backtracing to update
			 * velocities at each point on the lattice. Some 
			 * extra consideration when backtracing goes beyond
			 * simulation boundaries */

	#define TLTracedPosIdx (y-dQuadsY-1+yOff)*simWidth + (x-dQuadsX+xOff)
	#define TRTracedPosIdx (y-dQuadsY-1+yOff)*simWidth + (x-dQuadsX+1+xOff)
	#define BLTracedPosIdx (y-dQuadsY+yOff)*simWidth + (x-dQuadsX+xOff)
	#define BRTracedPosIdx (y-dQuadsY+yOff)*simWidth + (x-dQuadsX+1+xOff)

__device__ __inline__ bool
checkPosIdx (int posIdx,
			 unsigned int simWidth,
			 unsigned int simHeight)
{
	if(posIdx > simWidth*simHeight-1 || posIdx < 0){
		printf("Went out of bounds to: %i \n", posIdx);
		return false;
	}
	return true;
}

__global__ void 
Advect(float2 *__restrict__ positions,
	   float2 *__restrict__ oldVel, 
	   float2 *__restrict__ newVel,
	   float dt,
	   float dr,
	   float4 boundaries,
	   unsigned int simWidth,
	   unsigned int simHeight,
	   unsigned int testX,
	   unsigned int testY,
	   bool test)
{
		// actual realPos index
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
		// offsets to determine neighbors of interpolation with dependance on direction
	int xOff;
	int yOff;
	/*
	if(x==simWidth/4 && y==simWidth/2){
		newVel[y*simWidth+x] = make_float2(0.0, 0.0);
		return;
	}*/
	if(y*simWidth+x >= simWidth*simHeight){
		printf("Tried to access lattice position outside of memory \n");
		return;}
	
	float2 tracedPos;

	float dx = oldVel[y*simWidth+x].x * dt;
	float dy = oldVel[y*simWidth+x].y * dt;
	if (dx >= 0){ xOff = -1; } else xOff = 0;
	if (dy >= 0){ yOff = 1;} else yOff = 0;

	if(y==testY && x==testX && test==true){
		printf("dx : %f  - dy : %f\n", dx, dy);
	}

	tracedPos.x = positions[y*simWidth+x].x - oldVel[y*simWidth+x].x * dt;
	tracedPos.y = positions[y*simWidth+x].y - oldVel[y*simWidth+x].y * dt;
	if(y==testY && x==testX && test==true){
		printf("tracedPos.x : %f  - tracedPos.y : %f\n", tracedPos.x, tracedPos.y);
	}
	if(x==testX && y==testY && test==true){
		//printf("happeneddddddd");
	}

			// Top and Bottom held to zero as boundary condition
	
	if(y==0 || y==simHeight-1)
	{
		oldVel[y*simWidth+x] = make_float2(0.0, 0.0);
		return;
	}

				// change in realQuad position

	unsigned int dQuadsX = floor(dx / dr);
	unsigned int dQuadsY = floor(dy / dr);
	if(y==testY && x==testX && test==true){
		printf("dquadsX : %f  - dquadsY : %f\n", dQuadsX, dQuadsY);
	}
	if(y==testY && x==testX && test==true){
		printf("xOff : %i  - yOff : %f\n", xOff, yOff);
		printf("corners: %i %i %i %i\n", TLTracedPosIdx, TRTracedPosIdx, BLTracedPosIdx, BRTracedPosIdx);	
	}


			// is tracedPos within simulation boundaries

	if(tracedPos.x > boundaries.x && 
	   tracedPos.x < boundaries.z &&
	   tracedPos.y > (boundaries.y+dr) &&
	   tracedPos.y < boundaries.w) {
	   	if( BRTracedPosIdx >= simWidth*simHeight ){
			printf("Traced to a quadIdx that was out of bounds: %i \n", 
				  (y-dQuadsY+1)*simWidth+(x-dQuadsX));
			return;
		}

		if(y==testY && x==testX && test==true){
			printf("Tracing from: %f, %f \n", positions[y*simWidth+x].x, positions[y*simWidth+x].y);
			printf("Got traced Position: %f, %f \n", tracedPos.x, tracedPos.y);
			printf("With original velocities: %f, %f \n", oldVel[y*simWidth+x].x, oldVel[y*simWidth+x].y);
			printf("dQuads x and y: %i, %i \n", dQuadsX, dQuadsY);
		}

		float2 TLPos = positions[ TLTracedPosIdx ];
		float2 TRPos = positions[ TRTracedPosIdx ];
		float2 BLPos = positions[ BLTracedPosIdx ];
		float2 BRPos = positions[ BRTracedPosIdx];
		if(y==testY && x==testX && test==true){
			printf("Interpolating between velocities at positions: \n %f, %f \n %f, %f \n %f, %f \n %f, %f \n", TLPos.x, TLPos.y, BLPos.x, BLPos.y, BRPos.x, BRPos.y, TRPos.x, TRPos.y);
		}

		float2 TLVel = oldVel[ TLTracedPosIdx ];
		float2 TRVel = oldVel[ TRTracedPosIdx ];
		float2 BLVel = oldVel[ BLTracedPosIdx ];
		float2 BRVel = oldVel[ BRTracedPosIdx ];
		if(y==testY && x==testX && test==true){
			printf("And Velocities: \n %f, %f \n %f, %f \n %f, %f \n %f, %f \n", TLVel.x, TLVel.y, BLVel.x, BLVel.y, BRVel.x, BRVel.y, TRVel.x, TRVel.y);
		}

		if(tracedPos.y == TLPos.y){
			newVel[y*simWidth+x] = LinInterp(tracedPos,
											 TLVel, TLPos, 
											 TRVel, TRPos,
											 dr);
			
			if(y==testY && x==testX && test==true)
				printf("Velocity became from LININTERP: %f %f \n\n", newVel[y*simWidth+x].x, newVel[y*simWidth+x].y);
			

			return;
		}

		newVel[y*simWidth+x] = BiLinInterp(tracedPos, 
										   TLVel, TLPos,
										   BLVel, BLPos,
										   BRVel, BRPos,
										   TRVel, TRPos,
										   dr);
		if(y==testY && x==testX && test==true)
			printf("Velocity became from BILININTERP: %f %f \n\n", newVel[y*simWidth+x].x, newVel[y*simWidth+x].y);

		return;
		/*
		if(y*simWidth+x == simWidth*simHeight-1)
			printf("Final lattice point is doing things \n\n");
		*/
	}
		// Traced Position beyond LEFT hand boundary (x)

	if(tracedPos.x < boundaries.x &&
	   tracedPos.x < boundaries.z)
	{
		newVel[y*simWidth+x] = oldVel[y*simWidth+x];
		//newVel[y*simWidth+x].x = 3.0;
		//newVel[y*simWidth+x].y = 0.0;
		return;
	}

		// Traced Position beyond RIGHT hand boundary (z)

	if(tracedPos.x > boundaries.x && 
	   tracedPos.x > boundaries.z)
	{
		//newVel[y*simWidth+x] = oldVel[y*simWidth+x];
		
		newVel[y*simWidth+x].x = -5.0;
		newVel[y*simWidth+x].y = 0.0;
		

		return;
	}

		// Traced Position beyond TOP boundary (w)
	
	if(tracedPos.y > boundaries.y && 
	   tracedPos.y >= boundaries.w && 
	   tracedPos.x > boundaries.x &&
	   tracedPos.x < boundaries.z)
	{

		//printf("top boundary indexes : %i %i \n",  TLTracedPosIdx, TRTracedPosIdx);
		if(checkPosIdx(TLTracedPosIdx, simWidth, simHeight) && 
		   checkPosIdx(TRTracedPosIdx, simWidth, simHeight)){

		float2 LPos = positions[ TLTracedPosIdx ];
		float2 RPos = positions[ TRTracedPosIdx ];

		float2 LVel = oldVel[ TLTracedPosIdx ];
		float2 RVel = oldVel[ TRTracedPosIdx ];

		newVel[y*simWidth+x] = LinInterp(tracedPos, 	
										 LVel, LPos,
										 RVel, RPos,
										 dr);
		}
		return;
	}

		// Traced Position beyond BOTTOM boundary (y)
	
	if(tracedPos.y < boundaries.w &&
	   tracedPos.y <= boundaries.y && 
	   tracedPos.x > boundaries.x &&
	   tracedPos.x < boundaries.z)
	{
		
		//printf("bottom boundary indexes : %i %i \n",  TLTracedPosIdx, TRTracedPosIdx);
		if(checkPosIdx(TLTracedPosIdx, simWidth, simHeight) && 
		   checkPosIdx(TRTracedPosIdx, simWidth, simHeight)){

		float2 LPos = positions[ TLTracedPosIdx ];
		float2 RPos = positions[ TRTracedPosIdx ];

		float2 LVel = oldVel[ TLTracedPosIdx ];
		float2 RVel = oldVel[ TRTracedPosIdx ];

		newVel[y*simWidth+x] = LinInterp(tracedPos, 	
										 LVel, LPos,
										 RVel, RPos,
										 dr);
		}
		return;
	}
}

			/* Update old velocity data to current velocity data.
			 * These update at random and so must be updated in 
			 * bulk after velocity calculations are done */

__global__ void 
updateVel(float2 *__restrict__ oldVel,
		  float2 *__restrict__ newVel,
		  unsigned int simWidth)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	oldVel[y*simWidth+x] = newVel[y*simWidth+x];
}

			/* Didn't want to write out all these multiple entries
			 * for float2 calculations every time */

__device__ float2
JacobiInstance(float2 Top, 
			   float2 Left,
			   float2 Bot,
			   float2 Right,
			   float Alpha,
			   float2 Val)
{
	float2 res;
	res.x = (Top.x + Left.x + Bot.x + Right.x + Alpha * Val.x) / (4 + Alpha);
	res.y = (Top.y + Left.y + Bot.y + Right.y + Alpha * Val.y) / (4 + Alpha);
	return res;
}

			/* Bilinear Interpolation of velocities at four nearest
			 * mesh points giving the expected velocity at an arbitrary
			 * point contained */

__device__ float2
BiLinInterp(float2 pos,
 		 	 float2 TLVel, float2 TLPos,
			 float2 BLVel, float2 BLPos,
			 float2 BRVel, float2 BRPos,
			 float2 TRVel, float2 TRPos,
			 float dr)
{
	float2 TopInterp;
	float2 BotInterp;
	TopInterp.x = (TRPos.x - pos.x)/(TRPos.x - TLPos.x)*TLVel.x + 
				  (pos.x - TLPos.x)/(TRPos.x - TLPos.x)*TRVel.x;

	BotInterp.x = (BRPos.x - pos.x)/(BRPos.x - BLPos.x)*BLVel.x + 
				  (pos.x - BLPos.x)/(BRPos.x - BLPos.x)*BRVel.x;
	
	TopInterp.y = (TRPos.x - pos.x)/(TRPos.x - TLPos.x)*TLVel.y + 
				  (pos.x - TLPos.x)/(TRPos.x - TLPos.x)*TRVel.y;

	BotInterp.y = (BRPos.x - pos.x)/(BRPos.x - BLPos.x)*BLVel.y + 
				  (pos.x - BLPos.x)/(BRPos.x - BLPos.x)*BRVel.y;
	
	float2 ResInterp;
	ResInterp.x = (TLPos.y - pos.y)/(TLPos.y - BRPos.y)*TopInterp.x +
				  (pos.y - BRPos.y)/(TLPos.y - BRPos.y)*BotInterp.x;

	ResInterp.y = (TLPos.y - pos.y)/(TLPos.y - BRPos.y)*TopInterp.y + 
				  (pos.y - BRPos.y)/(TLPos.y - BRPos.y)*BotInterp.y;
	
	return ResInterp;
}

			/* Linear interpolation between velocities along an edge, 
			 * between two points. Used for top/bottom edge cases where
			 * otherwise bilinear interpolation would want to read outside
			 * of data bounds */

__device__ float2
LinInterp(float2 pos,
			 float2 LVel, float2 LPos,
			 float2 RVel, float2 RPos,
			 float dr)
{
	float2 interp;
	interp.x = (RPos.x - pos.x)/(RPos.x - LPos.x)*LVel.x + 
			   (pos.x - LPos.x)/(RPos.x - LPos.x)*RVel.x;

	interp.y = (RPos.x - pos.x)/(RPos.x - LPos.x)*LVel.y + 
			   (pos.x - LPos.x)/(RPos.x - LPos.x)*RVel.y;

	return interp;
}

void runCuda(struct hipGraphicsResource **vboResource,
			 int *obstructed,
			 float3 *colorMap,
			 float2 *pos,
			 float2 *oldVel,
			 float2 *newVel,
			 float4 boundaries,
			 float dt,
			 float dr,
			 dim3 tpbColor,
			 dim3 tpbLattice,
			 dim3 blocks,
			 unsigned int simWidth,
			 unsigned int simHeight, 
			 unsigned int testX, unsigned int testY, bool test)
{
	float3 *devPtr;
	checkCuda(hipGraphicsMapResources(1, vboResource, 0));
	size_t numBytes;
	checkCuda(hipGraphicsResourceGetMappedPointer((void**)&devPtr, &numBytes,
												   *vboResource));

	Obstruct<<<1, 16>>>(obstructed, oldVel);
	
	Advect<<< blocks, tpbLattice >>>(pos, oldVel, newVel, 
									 dt, dr, boundaries,
									 simWidth, simHeight, testX, testY, test);
	checkCuda(hipPeekAtLastError());
	checkCuda(hipDeviceSynchronize());

	updateVel<<< blocks, tpbLattice >>>(oldVel, newVel, simWidth);
	checkCuda(hipDeviceSynchronize());

	
	float viscosity = 1.48e-5;
	for (int i = 0; i < 40; i++){
		Diffuse<<< blocks, tpbLattice >>> (pos, oldVel, newVel, 
										   dt, dr, viscosity,
										   simWidth, simHeight);
		checkCuda(hipPeekAtLastError());
		updateVel<<< blocks, tpbLattice >>>(oldVel, newVel, simWidth);
		//std::cout<<"	Running Jacobi Diffusion: "<<i<<std::endl;
	}



	updateVel<<< blocks, tpbLattice >>>(oldVel, newVel, simWidth);

	velToColor<<< blocks, tpbLattice >>>(devPtr, colorMap, oldVel, blocks, simWidth, simHeight);
	checkCuda(hipDeviceSynchronize());

	checkCuda(hipGraphicsUnmapResources(1, vboResource, 0));
}

void glfwInitContext(GLFWwindow* window)
{
	if (!glfwInit()) {
		fprintf(stderr, "ERROR: could not start GLFW3\n");
		exit(1);
	} 

	window = glfwCreateWindow(1920/2, 1080/2, "Cuda Fluid Dynamics", NULL, NULL);
	glfwSetWindowPos(window, 1920/2, 0);
	if (!window) {
		fprintf(stderr, "ERROR: could not open window with GLFW3\n");
		glfwTerminate();
		exit(1);
	}
	glfwMakeContextCurrent(window);

	glewExperimental = GL_TRUE;
	glewInit();
}

void glInitShaders(const char *vertexShaderText,
			 	   const char *fragmentShaderText,
				   GLuint shaderProgram)
{
	GLuint vertexShader = glCreateShader(GL_VERTEX_SHADER);
	glShaderSource(vertexShader, 1, &vertexShaderText, NULL);
	glCompileShader(vertexShader);
	int params = -1;
	glGetShaderiv(vertexShader, GL_COMPILE_STATUS, &params);
	if(GL_TRUE != params){	
		int actual_length = 0;
		char log[2048];
		fprintf(stderr, "ERROR: GL shader idx %i did not compile\n", vertexShader);
		glGetShaderInfoLog(vertexShader, 500, &actual_length, log);
		std::cout << log;
		exit(1);
	}
	GLuint fragmentShader = glCreateShader(GL_FRAGMENT_SHADER);
	glShaderSource(fragmentShader, 1, &fragmentShaderText, NULL);
	glCompileShader(fragmentShader);
	glGetShaderiv(fragmentShader, GL_COMPILE_STATUS, &params);
	if(GL_TRUE != params){	
		int actual_length = 0;
		char log[2048];
		fprintf(stderr, "ERROR: GL shader idx %i did not compile\n", vertexShader);
		glGetShaderInfoLog(vertexShader, 500, &actual_length, log);
		std::cout << log;
		exit(1);
	}
	glAttachShader(shaderProgram, vertexShader);
	glAttachShader(shaderProgram, fragmentShader);
	glLinkProgram(shaderProgram);
}

void initThreadDimensions(unsigned int simWidth,
						  unsigned int simHeight,
						  dim3 &tpbColor,
						  dim3 &tpbLattice,
						  dim3 &blocks)
{
	int xBlocks;
	int yBlocks;
	int numThreads = simWidth*4*simHeight;	
	switch(numThreads){
		case 1024:
			xBlocks = 1;
			yBlocks = 8;
			break;
		case 4096:
			xBlocks = 1;
			yBlocks = 8;
			break;
		case 16384:
			xBlocks = 1;
			yBlocks = 32;
			break;
		case 65536:
			xBlocks = 1;
			yBlocks = 128;
			break;
		case 262144:
			xBlocks = 2;
			yBlocks = 256;
			break;
		default:
			std::cout<<"Bad Dimensions"<<std::endl;
			exit(1);
	}
			
	tpbColor.x = simWidth*4/xBlocks;
	tpbColor.y = simHeight/yBlocks;
	tpbLattice.x = simWidth/xBlocks;
	tpbLattice.y = simWidth/yBlocks;
	blocks.x = xBlocks;
	blocks.y = yBlocks;
	std::cout<<"	Calling kernels with:"<<std::endl
			 <<"	ThreadsPerBlock: ["<<tpbLattice.x<<", "<<tpbLattice.y<<"]"<<std::endl
			 <<"	On a Grid of: ["<<blocks.x<<"x"<<blocks.y<<"] Blocks"<<std::endl;
}


